#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <assert.h>

#define BLOCK_SIZE 16


__global__ void gpu_matrix_mult(int *a,int *b, int *c, int m, int n, int k)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int sum = 0;
    if( col < k && row < m)
    {
        for(int i = 0; i < n; i++)
        {
            sum += a[row * n + i] * b[i * k + col];
        }
        c[row * k + col] = sum;
    }
}



__global__ void gpu_matrix_transpose(int* mat_in, int* mat_out, unsigned int rows, unsigned int cols)
{
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int idy = blockIdx.y * blockDim.y + threadIdx.y;

    if (idx < cols && idy < rows)
    {
        unsigned int pos = idy * cols + idx;
        unsigned int trans_pos = idx * rows + idy;
        mat_out[trans_pos] = mat_in[pos];
    }
}



int main(int argc, char const *argv[])
{
    int m=2;
    int n=2;
    int k=2;
    int *h_a, *h_b, *h_c, *h_cc;
    hipHostMalloc((void **) &h_a, sizeof(int)*m*n, hipHostMallocDefault);
    hipHostMalloc((void **) &h_b, sizeof(int)*n*k, hipHostMallocDefault);
    hipHostMalloc((void **) &h_c, sizeof(int)*m*k, hipHostMallocDefault);
    hipHostMalloc((void **) &h_cc, sizeof(int)*m*k, hipHostMallocDefault);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);
    printf("matrix A\n");
    for (int i = 0; i < m; ++i) {
        for (int j = 0; j < k; ++j) {
            h_a[i * n + j] = rand()%10+1;
            printf( " %d \t" , h_a[i * n + j] ) ;
        }
        printf("\n");
    }
     printf("\nmatrix B\n");
    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < k; ++j) {
            h_b[i * k + j] = rand()%10+1;
            printf( " %d \t" , h_b[i * k + j] ) ;
        }
        printf("\n");
    }


    int *d_a, *d_b, *d_c;
    hipMalloc((void **) &d_a, sizeof(int)*m*n);
    hipMalloc((void **) &d_b, sizeof(int)*n*k);
    hipMalloc((void **) &d_c, sizeof(int)*m*k);

    hipMemcpy(d_a, h_a, sizeof(int)*m*n, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, sizeof(int)*n*k, hipMemcpyHostToDevice);

    unsigned int grid_rows = (m + BLOCK_SIZE - 1) / BLOCK_SIZE;
    unsigned int grid_cols = (k + BLOCK_SIZE - 1) / BLOCK_SIZE;
    dim3 dimGrid(grid_cols, grid_rows);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

    
        gpu_matrix_mult<<<dimGrid, dimBlock>>>(d_a, d_b, d_c, m, n, k);
    
    hipMemcpy(h_c, d_c, sizeof(int)*m*k, hipMemcpyDeviceToHost);


    printf("Result of multiplying\n");
   for(int i = 0; i < m; i++)
     {
        for(int j = 0; j < n; j++)
        {
            printf( " %d \t" , h_c[i+j] ) ;
        }
        printf("\n");
     }

    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("The time is %.6f ms\n", milliseconds);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hipHostFree(h_a);
    hipHostFree(h_b);
    hipHostFree(h_c);
    hipHostFree(h_cc);
    return 0;
}

