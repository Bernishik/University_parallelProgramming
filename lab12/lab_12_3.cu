#pragma comment (lib, "cublas.lib")
#pragma comment (lib, "curand.lib")


#include <hip/hip_runtime.h>
#include <cstdlib>
#include <hiprand/hiprand.h>
#include <stdio.h>
#include <stdlib.h>
#include "time.h"
#include <hipblas.h>

void GPU_fill_rand(float *A, int nr_rows_A, int nr_cols_A) {
    hiprandGenerator_t prng;
    hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(prng, time(0));
    hiprandGenerateUniform(prng, A, nr_rows_A * nr_cols_A);
}

void gpu_blas_mmul(const float *A, const float *B, float *C, const int m, const int k, const int n) {
    int lda = m, ldb = k, ldc = m;
    const float alf = 1;
    const float bet = 0;
    const float *alpha = &alf;
    const float *beta = &bet;
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);
    hipblasDestroy(handle);
}

void print_matrix(const float *A, int nr_rows_A, int nr_cols_A) {
    for (int i = 0; i < nr_rows_A; ++i) {
        for (int j = 0; j < nr_cols_A; ++j) {
            printf("%f  ", A[j * nr_rows_A + i]);
        }
        printf("\n");
    }
    printf("\n");
}

int main() {
    int m, k, n;
    m = 10000;
    k = 10000;
    n = 10000;
    float *h_A = (float *) malloc(m * k * sizeof(float));
    float *h_B = (float *) malloc(n * k * sizeof(float));
    float *h_C = (float *) malloc(m * n * sizeof(float));

    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, m * k * sizeof(float));
    hipMalloc(&d_B, n * k * sizeof(float));
    hipMalloc(&d_C, m * n * sizeof(float));

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);
    GPU_fill_rand(d_A, m, k);
    GPU_fill_rand(d_B, n, k);

    hipMemcpy(h_A, d_A, m * k * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(h_B, d_B, n * k * sizeof(float), hipMemcpyDeviceToHost);
//    printf("A:\n");
//    print_matrix(h_A, m, k);
//    printf("B:\n");
//    print_matrix(h_B, n, k);

    gpu_blas_mmul(d_A, d_B, d_C, m, k, k);

    hipMemcpy(h_C, d_C, m * n * sizeof(float), hipMemcpyDeviceToHost);
//    printf("C:\n");
//    print_matrix(h_C, m, n);
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("The time is %.6f ms\n", milliseconds);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    free(h_A);
    free(h_B);
    free(h_C);
    return 0;
}

// nvcc -o lab12_3 lab_12_3.cu -lcurand -lcublas

