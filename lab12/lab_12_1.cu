#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
int main()
{
    const int m=100;
    const int k=100;
    const int n=100;
    int a[m][k];
    int b[k][n];
    int c[m][n];
    int i,j,q;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start,0);
//    printf("matrix A\n");
     for( i = 0; i < m; i++)
        {
             for( j = 0; j < k; j++)
             {
                a[i][j]=rand()%10+1;
//                printf( " %d \t" , a[i][j] ) ;
             }
//              printf("\n");
        }
//    printf("matrix B\n");
        for( i = 0; i < k; i++)
        {
             for( j = 0; j < n; j++)
             {
                b[i][j]=rand()%10+1;
//                printf( " %d \t" , b[i][j] ) ;
             }
//             printf("\n");
        }
//    printf("Result of multiplying\n");
     for(i = 0; i < m; i++)
     {
        for(j = 0; j < n; j++)
        {
            c[i][j] = 0;
            for(q = 0; q < k; q++)
                c[i][j] += a[i][q] * b[q][j];
//            printf( " %d \t" , c[i][j] ) ;
        }
//        printf("\n");
     }
     hipEventRecord(stop,0);
     hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("The time is %.6f ms\n", milliseconds);
    return 0;
}

