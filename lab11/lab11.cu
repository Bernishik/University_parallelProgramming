#include <stdio.h>

int main() {
  int nDevices;

  hipGetDeviceCount(&nDevices);
  for (int i = 0; i < nDevices; i++) {
      hipDeviceProp_t prop;
      hipGetDeviceProperties(&prop, i);
      printf("Device Number: %d\n", i); //Номер девайсу
      printf("  Device name: %s\n", prop.name); //Назва девайсу:
      printf("Major cap: %d\n",prop.major); // Основна версія (обчислювальних можливостей)
      printf("Minor cap: %d\n",prop.minor); // Вторинна версія (обчислювальних можливостей)
      printf("  Memory Clock Rate (KHz): %d\n",prop.memoryClockRate); // Тактова частота
      printf("Global memory %zu\n",prop.totalGlobalMem); // Глобальна память
      printf("Constant memory %d\n",int(prop.totalConstMem)); // Память констант
      printf("Shared memory %d\n",int(prop.sharedMemPerMultiprocessor)); //Спільна память
      printf("Multiproccessors %d\n",prop.multiProcessorCount); // Кількість мультипроцессорів
      printf("Threads in block %d\n",prop.maxThreadsPerBlock); //Максимальна кількість потоків у блоці
      printf("Threads size %d\t %d\t %d\n",prop.maxThreadsDim[0],prop.maxThreadsDim[1],prop.maxThreadsDim[2]); //Розмірність потоку
      printf("Grid  size %d\t %d\t  %d\n",prop.maxGridSize[0],prop.maxGridSize[1],prop.maxGridSize[2]); //Розмірність Сітки
      printf("Single to double precision %d\n",prop.singleToDoublePrecisionPerfRatio);//Співвідношення еффективності одиничної та подвійної точності

  }


}
